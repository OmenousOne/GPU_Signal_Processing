#include "gpu_type.h"
#include <iostream>
#include <cstring>
#include <hip/hip_runtime.h>
#include <npp.h>
#include <sys/time.h>

#define EHEAD 3200
#define BHEAD 400
#define THEAD 240

// globals declared here
int32  netype,nbtype,nformat;
int32  iendian=-1;
int32  iswapd=0;
int32  ntrac,ntaux,nsamp,nsint,nrev,nrevd,nbpsamp,ntraces;
int64  lfilesz=0;
timeval tStart;

int32 main(int nargc, char *Cargv[])
{
   // declare variables
   int32  i=0,isize,numproc=0;
   int32  nread=0,nwrite=0,numt=0;
   int64  loc=0;
   float  fscale=0.0f;
   float  *Fdata,*cu_Fdata;
   char   Cin[256],Cout[256];
   char   *Cdata;
   char   Cehead[EHEAD],Cbhead[BHEAD];  // binary and ebcdic headers
   FILE   *Fin=NULL,*Fout=NULL;
   

   // determine endian of machine   
   iendian = endian(1);

   // usage information
   if (nargc<=1)
   {
      std::cout << "Usage:  gpu_signal_processing.exe infile outfile multiplier traces" << std::endl;
      std::cout << "" << std::endl;
      std::cout << "        infile      - SEGY IEEE 4byte Floating Point File, big or little endian" << std::endl;
      std::cout << "        outfile     - Signal Processed SEGY IEEE 4byte Floating Point File, same endian as input" << std::endl;
      std::cout << "        multiplier  - float to multiply samples by" << std::endl;
      std::cout << "        traces      - number of traces to work on simultaneously" << std::endl;
      std::cout << "" << std::endl;
      std::cout << "        example: gpu_signal_processing.exe ../data/sample_LE.sgy output.sgy 2.0 4" << std::endl;
      std::cout << "" << std::endl;
      return(1);
   }

   // parse command line args
   for(i=1; i<nargc; i++)
   {
      if (nargc<5)
      {  
         std::cout << "err - missing argument(s)" << std::endl;
         std::cout << "      run with no arguments to view usage information" << std::endl;
         std::cout << "" << std::endl;
         return(-1);
      }
      if (i==1)  
      {
         strcpy(Cin,Cargv[i]);
         std::cout << "Input File: " << Cin << std::endl;
      } 
      else if (i==2)  
      {
         strcpy(Cout,Cargv[i]);
         std::cout << "Output File: " << Cout << std::endl;
      } 
      else if (i==3)  
      {
         fscale = atof(Cargv[i]);
         std::cout << "Multiplier: " << fscale << std::endl;
      } 
      else if (i==4)  
      {
         numt = atoi(Cargv[i]);
         std::cout << "Simultaneous Traces: " << numt << std::endl;
      } 
   }
   std::cout << "" << std::endl;

   // start timing
   etime();

   // open input file
   Fin = fopen(Cin,"rb");   // open for read binary 
   if (!Fin)
   {
      std::cout << "err - failed to open for read: " << Cin << std::endl;
      return(0);
   }
   std::cout << "Open for read: " << Cin << std::endl;
   // get file size
   lfilesz = fseeko(Fin,0,SEEK_END);
   lfilesz = ftello(Fin);
   rewind(Fin);


   // test if it is SEGY data
   nread=fread(Cehead,1,EHEAD,Fin);
   if (nread!=EHEAD)
   {
      std::cout << "err - Read " << nread << "Bytes, expected " << EHEAD << std::endl;
      return(-1);
   }
   std::cout << "EBCDIC Header Read " << nread << " Bytes" << std::endl; 
   nread=fread(Cbhead,1,BHEAD,Fin);
   if (nread!=BHEAD)
   {
      std::cout << "err - Read " << nread << "Bytes, expected " << BHEAD << std::endl;
      return(-1);
   }
   std::cout << "Binary Header Read " << nread << " Bytes" << std::endl; 

   // test if it is a supported SEGY type
   if (segy_check(&Cehead[0],&Cbhead[0]) != 0)
   {
      std::cout << "err - not a valid/supported SEGY type data" << std::endl;
      return(-1);
   }

   // data type okay so prepare output file with headers
   // open output file
   Fout = fopen(Cout,"wb");   // open for write binary 
   if (!Fout)
   {
      std::cout << "err - failed to open for write: " << Cout << std::endl;
      return(0);
   }
   std::cout << "Open for write: " << Cout << std::endl;
   nwrite=fwrite(Cehead,1,EHEAD,Fout);
   if (nwrite!=EHEAD)
   {
      std::cout << "err - Wrote " << nwrite << "Bytes, expected " << EHEAD << std::endl;
      return(-1);
   }
   nwrite=fwrite(Cbhead,1,BHEAD,Fout);
   if (nwrite!=BHEAD)
   {
      std::cout << "err - Wrote " << nwrite << "Bytes, expected " << BHEAD << std::endl;
      return(-1);
   }
   
   // set variables and allocate buffers to hold traces 
   // numt=1;
   // fscale=2.0f;
   isize=nsamp*sizeof(float);
   Cdata=(char*)calloc((numt*(isize+THEAD)),1);    // traces including header
   Fdata=(float*)calloc((numt*isize),1);         // just float samples from traces
   hipMalloc((void**)&cu_Fdata,numt*isize);

   ///////////////////////////////////////////////
   //
   // loop through data until all file processed
   //
   ///////////////////////////////////////////////
   for (loc=3600; loc<lfilesz; )
   {
      // read in trace data, should be either big or little endian ieee floating point 4 byte data
      nread = read_trace(Cdata,numt,Fin);
      if (nread<0)  return(-1); 
      if (nread==0) break;                 // found end of file
      loc+=nread*(isize+THEAD);  // set location in file
 
      // transfer trace data to float buffer
      trace_2floatbuff(Cdata,Fdata,nread);
      if (numproc==0)
      {
         std::cout << "" << std::endl;
         std::cout << "First 8 data samples for QC:" << std::endl;
         // output some before values
         for(i=0; i<8; i++)
         {
            printf("Input %d: %0.8f\n",i,Fdata[i]);
         }
      }
      // copy to device memory
      hipMemcpy(cu_Fdata,Fdata,numt*isize,hipMemcpyHostToDevice);

      // apply GPU signal processing
      hipDeviceSynchronize();
      nppsMulC_32f_I(fscale,cu_Fdata,numt*isize);
      hipDeviceSynchronize();

      // copy to host memory
      hipMemcpy(Fdata,cu_Fdata,numt*isize,hipMemcpyDeviceToHost);
      if (numproc==0)
      {
         // output some after values
         std::cout << "" << std::endl;
         for(i=0; i<8; i++)
         {
            printf("Output %d: %0.8f\n",i,Fdata[i]);
         }
         std::cout << "" << std::endl;
      }
      // transfer trace data to float buffer
      float_2tracebuff(Fdata,Cdata,nread);

      // write out data to output file
      nwrite = write_trace(Cdata,nread,Fout);
      if (nwrite<=0)  break; 
      numproc+=nwrite;
   }

   // free memory
   hipFree(cu_Fdata);
   free(Cdata);
   free(Fdata);

   // close files
   if (Fin!=NULL)  fclose(Fin);
   if (Fout!=NULL) fclose(Fout);

   std::cout << "Total Traces Processed: "<< numproc << std::endl;
   std::cout << "Total Samples Processed (float numbers): "<< numproc*nsamp << std::endl;
   std::cout << "Elapsed Time: " << etime() << " microseconds\n";

   return(0);
}


////////////////////////////////////////
//
// functions below here 
//
////////////////////////////////////////

void   trace_2floatbuff(char *Cbuff,float *Fbuff,int32 numt)
{
   int32 i=0,j=0;
   int32 icloc=0,ifloc=0;
   float *Fp;
   
   for (i=0; i<numt; i++)
   {
      icloc=i*((nbpsamp*nsamp)+THEAD)+THEAD;
      Fp=(float*)&Cbuff[icloc];
      for(j=0; j<nsamp; j++)
      {
         Fbuff[ifloc] = *Fp;
         ifloc++;
         Fp++;
      }
   }
   
   return;
}

void   float_2tracebuff(float *Fbuff,char *Cbuff,int32 numt)
{
   int32 i=0,j=0;
   int32 icloc=0,ifloc=0;
   float *Fp;
   
   for (i=0; i<numt; i++)
   {
      icloc=i*((nbpsamp*nsamp)+THEAD)+THEAD;
      Fp=(float*)&Cbuff[icloc];
      for(j=0; j<nsamp; j++)
      {
         *Fp= Fbuff[ifloc];
         ifloc++;
         Fp++;
      }
   }
   
   return;
}

int32 read_trace(char *Cbuff,int32 numt,FILE *Fi)
{
   // read trace(s) into buffer
   int32 nsize=0;
   int32 nread=0;
   int32 i=0,j=0,iloc=0;
   float *Fp;
   
   // read number of traces * trace size 
   //                         ((bytes per sample * number of samples )+trace header)
   nsize = numt * ((nbpsamp*nsamp)+THEAD);
   nread = fread(Cbuff,1,nsize,Fi);
   if (nread==0)
   {
      std::cout << "EOF - end of file, 0 traces read" << std::endl;
      return(0);
   }
   // convert to traces read
   nread = nread/((nbpsamp*nsamp)+THEAD);
   if (nread < 1)
   {
      std::cout << "err - less than a full trace read" << std::endl;
      return(-1);
   }

   // byte swap data if needed
   if (iswapd)
   {
      for(i=0; i<nread; i++)
      {
         // only byte swap data not headers
         iloc=i*(((nbpsamp*nsamp)+THEAD))+THEAD;
         Fp=(float*)&Cbuff[iloc];
         for(j=0; j<nsamp; j++)
         {
            *Fp = bswapf(*Fp);
            Fp++;  // move pointer to next sample
         }
      }
   }

   return(nread);
}

int32 write_trace(char *Cbuff,int32 numt,FILE *Fo)
{
   // write trace(s) to FILE
   int32 nsize=0;
   int32 nwrite=0;
   int32 i=0,j=0,iloc=0;
   float *Fp;
   
   // byte swap data if needed
   if (iswapd)
   {
      for(i=0; i<numt; i++)
      {
         // only byte swap data not headers
         iloc=(i*((nbpsamp*nsamp)+THEAD))+THEAD;
         Fp=(float*)&Cbuff[iloc];
         for(j=0; j<nsamp; j++)
         {
            *Fp = bswapf(*Fp);
            Fp++;  // move pointer to next sample
         }
      }
   }

   // write number of traces * trace size 
   //                         ((bytes per sample * number of samples )+trace header)
   nsize = numt * ((nbpsamp*nsamp)+THEAD);
   nwrite = fwrite(Cbuff,1,nsize,Fo);
   if (nwrite==0)
   {
      std::cout << "err - wrote 0 traces, disk full?" << std::endl;
      return(0);
   }
   // convert to traces read
   nwrite = nwrite/((nbpsamp*nsamp)+THEAD);
   if (nwrite != numt)
   {
      std::cout << "err - Wrote " << nwrite << "Traces, expected " << numt << std::endl;
      return(-1);
   }

   return(nwrite);
}

int32 segy_check(char *Ce, char *Cb)
{
   // check if valid/supported segy type data
   int32  i,na=0,ne=0;
   int32  *Lp;
   int16  *Sp;
   char   ch;

   for (i=0; i<180; i++)
   {
      ch=Ce[i];
      if (ch == '\x40') ne++;
      if (ch == '\x20') na++;
   }
   if (na+ne>0)
   {
      netype=0;
      if (ne>na) netype=1;
      if(netype) std::cout << "Ebcdic Header: EBCDIC" << std::endl;
      else       std::cout << "Ebcdic Header: ASCII" << std::endl;
   }
   else
   {
      std::cout << "Ebcdic Header: Type Not Found" << std::endl;
      return(-1); 
   }
   // grab info from binary header
   // revision
   Lp = (int32*)&Cb[96];
   nrev = *Lp;
   if ((!iendian && nbtype) || (iendian && !nbtype)) nrev = bswaps(nrev);
   if (nrev==16909060)
   {
      // this data sets endian matches this CPU
      std::cout << "Revision >= 2.0 Detected, Endian is correct for this CPU" << std::endl;
      nrevd=2;
   }
   else if(nrev==67305985)
   {
      // this data sets endian does NOT match this CPU
      std::cout << "Revision >= 2.0 Detected, Endian is NOT correct for this CPU" << std::endl;
      nrevd=-2;
   }
   else
   {
      std::cout << "Revision < 2.0 Detected" << std::endl;
      nrevd=0;
   }

   // format code
   Sp = (int16*)&Cb[24];
   nformat = *Sp;
   if (! iendian) nformat = bswaps(nformat); 
   if (nformat>255)
   {
      nformat = *Sp;
      nbtype=0;
      std::cout << "Binary Header: PC ORDER" << std::endl;
      
      if ((!iendian && nbtype) || (iendian && !nbtype)) nformat = bswaps(nformat);
   }
   else
   {
      nbtype=1;
      std::cout << "Binary Header: IBM ORDER" << std::endl;
   }
   if (nformat>20)
   {
      std::cout << "err - " << nformat << " is not a known format" << std::endl;
      return(-1);
   }
   if ((nformat!=6 && nformat!=11 && nrevd!=0 ) && (nformat!=5 && nrevd!=2 && nrevd!=-2))
   {
      std::cout << "err - Format: " << nformat << " Revision: " << nrevd << " is not supported currently" << std::endl;
      return(-1);
   }
   //assuming IEEE 4 bytes Floats
   nbpsamp=sizeof(float);

   // get some information about this data
   Sp = (int16*)&Cb[12];
   ntrac = *Sp;
   if ((!iendian && nbtype) || (iendian && !nbtype)) ntrac = bswaps(ntrac);
   Sp = (int16*)&Cb[14];
   ntaux = *Sp;
   if ((!iendian && nbtype) || (iendian && !nbtype)) ntaux = bswaps(ntaux);
   Sp = (int16*)&Cb[16];
   nsint = *Sp;
   if ((!iendian && nbtype) || (iendian && !nbtype)) nsint = bswaps(nsint);
   Sp = (int16*)&Cb[20];
   nsamp = *Sp;
   if ((!iendian && nbtype) || (iendian && !nbtype)) nsamp = bswaps(nsamp);
   
   printf("Num Traces:      %4d (%04X)  bytes 3213-3214\n",ntrac,ntrac);
   printf("Num Aux Tr:      %4d (%04X)  bytes 3215-3216\n",ntaux,ntaux);
   printf("Sample Interval: %4d (%04X)  bytes 3217-3218\n",nsint,nsint);
   printf("Num Samples:     %4d (%04X)  bytes 3221-3222\n",nsamp,nsamp);
   printf("Revision:  (%2d)  %4d (%04X)  bytes 3297-3300\n",nrevd,nrev,nrev);
   printf("Format:          %4d (%04X)  bytes 3225-3226\n\n",nformat,nformat);
   printf("Calculated Number of Traces:          %ld\n",(lfilesz-3600)/((nsamp*sizeof(float))+THEAD));

   if (nrevd==-2) iswapd=1;
   if (nrevd==0 && nformat==6 && iendian==0) iswapd=1;
   if (nrevd==0 && nformat==11 && iendian==1) iswapd=1;
   if (iswapd)  std::cout << "Data Byte Swap Required" << std::endl;

   return(0);
}
		
int32 etime(void) 
{
   timeval tEnd;
   int32   t;
 
   gettimeofday(&tEnd, 0);
   t = (tEnd.tv_sec - tStart.tv_sec) * 1000000 + tEnd.tv_usec - tStart.tv_usec;
   tStart = tEnd;
   return t;
}
